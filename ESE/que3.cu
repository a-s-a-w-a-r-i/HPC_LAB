/*------------------------------------------------------------------------------
  Program	: Write a program in CUDA to find factorial of a numbers.
  Programmer 	: Sandip Walunj  
  Designation 	: Assistant Professor, Computer Engineering Department,
                  SITRC, NAshik.
  Website 	: www.sandipwalunj.com
-------------------------------------------------------------------------------*/


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void Factorial(int *gpu_num,long int *gpu_res)
{
  int i;
  *gpu_res=1;
  for(i=1;i<=*gpu_num;i++)
  {
    *gpu_res = *gpu_res * i;      
  }
}

int main()
{
  int Number;  //to store number on the cpu/host machine
  int *dev_number;
  long int *res, result; //store result 
  system("clear"); //to clear the screen
  printf("\n\t Enter the number : ");
  scanf("%d",&Number);
  
  //to allocate memory for a number on the GPU/Device
  hipMalloc((void**)&dev_number,sizeof(int));
  hipMalloc((void**)&res,sizeof(long int));
  
  //copy number to the GPU/Device memory
  hipMemcpy(dev_number,&Number,sizeof(int),hipMemcpyHostToDevice);
 
  //call square function which will execute parallely on GPU
  Factorial<<<1,1>>>(dev_number,res);

  //copy result back from device/GPU back to CPU/Host
  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);

  //display result on the screen
  printf("\n\t Factorial of number %d is %ld \n",Number,result); 
 
  //deallocate GPU/Device memory
  return 0; 
}

/*
  Output:
  shri@shri-System-Product-Name:~/CUDA Programs$ nvcc Factorial.cu
  shri@shri-System-Product-Name:~/CUDA Programs$ ./a.out

	 Enter the number : 4   

	 Factorial of number 4 is 24 
  shri@shri-System-Product-Name:~/CUDA Programs$ 
*/
